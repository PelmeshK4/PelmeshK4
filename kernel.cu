#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <iomanip>



__global__ void kernel(double* arr, const double* a1, const double* a2, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	while (idx < n) {
		if (a1[idx] < a2[idx])
			arr[idx] = a1[idx];
		else arr[idx] = a2[idx];
		idx += offset;
	}
}


int main() {
	int i, n = 0;
	std::ios_base::sync_with_stdio(false);
	std::cin >> n;

	double* a1 = (double*)malloc(sizeof(double) * n);
	for (i = 0; i < n; i++) std::cin >> a1[i];
	double* dev_a1 = 0;
	hipMalloc((void**)&dev_a1, sizeof(double) * n);
	hipMemcpy(dev_a1, a1, sizeof(double) * n, hipMemcpyHostToDevice);
	double* a2 = (double*)malloc(sizeof(double) * n);
	for (i = 0; i < n; i++) std::cin >> a2[i];
	double* dev_a2 = 0;
	hipMalloc((void**)&dev_a2, sizeof(double) * n);
	hipMemcpy(dev_a2, a2, sizeof(double) * n, hipMemcpyHostToDevice);
	double* dev_arr = 0;
	hipMalloc((void**)&dev_arr, sizeof(double) * n);
	kernel << < 256, 256 >> > (dev_arr, dev_a1, dev_a2, n);
	double* arr = (double*)malloc(sizeof(double) * n);
	hipDeviceSynchronize();
	hipGetLastError();
	hipMemcpy(arr, dev_arr, sizeof(double) * n, hipMemcpyDeviceToHost);
	for (i = 0; i < n; i++)
		std::cout << std::setprecision(10) << std::scientific << arr[i] << "  ";
	hipFree(dev_a1);
	hipFree(dev_a2);
	hipFree(dev_arr);
	free(a1);
	free(a2);
	free(arr);
	return 0;
}
